#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <time.h>


#define N 512
void random_ints(int* a, int M)
{
        int i;
        for (i=0; i<M; i++) {
                a[i] = rand() % 5000;
        }
}
__global__ void add(int *a, int *b, int *c, int n) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if(index < n) c[index] = a[index] + b[index];
}
int main(void) {

        // host copies of a, b, c
        int *a, *b, *c;
        // device copies of a, b, c
        int *d_a, *d_b, *d_c;
        int size = N * sizeof(int);


        // Alloc space for device copies of a, b, c
        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);


        // Alloc space for host copies of a, b, c and setup input values
        a = (int *)malloc(size);
        random_ints(a, N);
        b = (int *)malloc(size);
        random_ints(b, N);
        c = (int *)malloc(size);

        // Copy inputs to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        // Launch add() kernel on GPU with N blocks
        add<<<2, N>>>(d_a, d_b, d_c, N);

        // Copy result back to host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

        for(int i=0; i<N; i++) {
                printf("%d", c[i]);
        }

        // Cleanup
        free(a); free(b); free(c);
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        return 0;
    }        